#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Calculator.h"
#include <cstdio>
#include <algorithm>

void CUDA::Grid3::GPUalloc() {
    if(isGPUalloc) {
        throw CUDA::Grid3GPUReallocEx();
    }

    hipMalloc((void**) &GPUdata, dataSize);

    isGPUalloc = true;
}

void CUDA::Grid3::GPUfree() {
    if(!isGPUalloc) {
        throw CUDA::Grid3GPUFreeEx();
    }

    hipFree(GPUdata);
}

void CUDA::Grid3::cpyDataFromGPU() {
    if(!isGPUalloc) {
        throw CUDA::Grid3WrongCallEx();
    }
    hipMemcpy(data, GPUdata, dataSize, hipMemcpyDeviceToHost);
}

void CUDA::Grid3::cpyDataToGPU() {
    if(!isGPUalloc) {
        throw CUDA::Grid3WrongCallEx();
    }
    hipMemcpy(GPUdata, data, dataSize, hipMemcpyHostToDevice);
}

void CUDA::Calculator::initTask() {
    hipMalloc((void**) &task, (param.SIZE - 2) * (param.SIZE - 2) * (param.SIZE_Z - 2) * sizeof(CUDA::Task));
    Task* _task = new Task[(param.SIZE - 2) * (param.SIZE - 2) * (param.SIZE_Z - 2)];
}

void CUDA::Calculator::freeTask() {
    hipFree(task);
}

__global__ void solve(double *data, double *goal, CUDA::Task* task) {
    int i = threadIdx.x + blockIdx.x * 512;

    CUDA::Task* cur = task + i;

    //printf("%d -> %d %d %d %d %d %d\n", cur->a, cur->a1, cur->a2, cur->a3, cur->a4, cur->a5, cur->a6);

    goal[cur->a] = (data[cur->a1] + data[cur->a2] + data[cur->a3] +
                    data[cur->a4] + data[cur->a5] + data[cur->a6]) / 6;

}


void CUDA::Calculator::calcU() {
	u->cpyDataToGPU();

	int N_OPERATION = param.SIZE * param.SIZE * param.SIZE_Z * log(1 / param.EPS);
	int boost = 1000;

    int n_tasks = param.SIZE * param.SIZE * param.SIZE_Z - borderSize;

    int counter = 0;

    for(int x = 1; x < param.SIZE - 1; x ++) {
	for(int y = 1; y < param.SIZE - 1; y ++) {
	for(int z = 1; z < param.SIZE_Z - 1; z ++) {

		if(border->at(x, y, z) == 0.0f) {
            _task[counter].a = border->p2n(x, y, z);
            _task[counter].a1 = border->p2n(x+1, y, z);
            _task[counter].a2 = border->p2n(x-1, y, z);
            _task[counter].a3 = border->p2n(x, y+1, z);
            _task[counter].a4 = border->p2n(x, y-1, z);
            _task[counter].a5 = border->p2n(x, y, z+1);
            _task[counter].a6 = border->p2n(x, y, z-1);

            counter++;
        }
	}}}

    hipMemcpy((void*) task, (void*)_task, n_tasks * sizeof(Task), hipMemcpyHostToDevice);

    std::cout << "tasks: " << n_tasks << " operations: " << N_OPERATION << std::endl;

    for(int op = 0; op < N_OPERATION / boost; op++) {
        solve<<< max(n_tasks / 512, 1), min(n_tasks, 512) >>>(u->getGPUdata(), prev_u->getGPUdata(), task);

        std::swap(u, prev_u);
    }

    hipDeviceSynchronize();
    u->cpyDataFromGPU();

}

template<class T>
__global__ void addKernel(T *c, const T *a, const T *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void CUDA::addInt(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipMalloc((void**) &dev_c, size * sizeof(int));
    hipMalloc((void**) &dev_a, size * sizeof(int));
    hipMalloc((void**) &dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<int><<<1 , size>>>(dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}