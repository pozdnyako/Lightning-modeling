#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Calculator.h"
#include <cstdio>
#include <algorithm>

template<class T>
__global__ void addKernel(T *c, const T *a, const T *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void CUDA::Grid3::GPUalloc() {
    if(isGPUalloc) {
        throw CUDA::Grid3GPUReallocEx();
    }

    hipMalloc((void**) &GPUdata, dataSize);

    isGPUalloc = true;
}

void CUDA::Grid3::GPUfree() {
    if(isGPUalloc) {
        throw CUDA::Grid3GPUFreeEx();
    }

    hipFree(GPUdata);
}

void CUDA::Grid3::cpyDataFromGPU() {
    if(!isGPUalloc) {
        throw CUDA::Grid3WrongCallEx();
    }
    hipMemcpy(GPUdata, data, dataSize, hipMemcpyDeviceToHost);
}

void CUDA::Grid3::cpyDataToGPU() {
    if(!isGPUalloc) {
        throw CUDA::Grid3WrongCallEx();
    }
    hipMemcpy(data, GPUdata, sizeof(double), hipMemcpyHostToDevice);
}

void CUDA::addInt(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipMalloc((void**) &dev_c, size * sizeof(int));
    hipMalloc((void**) &dev_a, size * sizeof(int));
    hipMalloc((void**) &dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<int><<<1, size>>>(dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}